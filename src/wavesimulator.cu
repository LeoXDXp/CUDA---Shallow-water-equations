#include "hip/hip_runtime.h"
#include "wavesimulator.h"

#include <iostream>
#include <glog/logging.h>
#include <gflags/gflags.h>

#include "types.h"
#include <stdlib.h>
#include <stdio.h>

#define BLOCKSIZE_X 8
#define BLOCKSIZE_Y 8

#ifndef min
#define min(a,b) (((a) < (b)) ? (a) : (b))
#endif

#ifndef max
#define max(a,b) (((a) > (b)) ? (a) : (b))
#endif

#define grid2Dwrite(array, x, y, pitch, value) array[(y)*pitch+x] = value
#define grid2Dread(array, x, y, pitch) array[(y)*pitch+x]

static bool validateTimestep(const char* flagname, double value)
{
    if (value > 0 && value < 1.0f)
        { return true; }
    std::cout << "Invalid value for --" << flagname << ": "
              << value << std::endl;
    return false;
}

static bool validateWSPF(const char* flagname, int value)
{
    if (value > 0)
        { return true; }
    std::cout << "Invalid value for --" << flagname << ": "
              << value << std::endl;
    return false;
}


DEFINE_int32(wspf, 50, "wavesteps per frame.");
DEFINE_double(timestep, 0.01f, "timestep between 2 wavesteps.");

static const bool timestep_dummy = google::RegisterFlagValidator(&FLAGS_timestep, &validateTimestep);
static const bool wspf_dummy = google::RegisterFlagValidator(&FLAGS_wspf, &validateWSPF);

const int UNINTIALISED = 0;
const int INITIALISED = 1;

int state = UNINTIALISED;

float timestep;
int stepsperframe = 50;

const float GRAVITY = 9.83219f * 0.5f; //0.5f * Fallbeschleunigung

const float NN = 5.0f;

texture<gridpoint, 2, hipReadModeElementType> texture_grid;
texture<vertex, 2, hipReadModeElementType> texture_landscape;

int grid_pitch_elements;
hipChannelFormatDesc grid_channeldesc;

gridpoint* device_grid;
gridpoint* device_grid_next;

vertex* device_heightmap;
vertex* device_watersurfacevertices;
float* device_treshholds;

float* device_waves;
rgb* device_watersurfacecolors;

#define EPSILON 0.0001f

__host__ __device__ gridpoint F(gridpoint gp)
{
    float h = gp.x;
    float uh = gp.y;
    float vh = gp.z;
    
    float h4 = h*h*h*h;
    float u = sqrtf(2)*h*uh/(sqrtf(h4 + max(h4, EPSILON)));

    gridpoint F;
    F.x = u*h;
    F.y = uh * u + GRAVITY * h * h;
    F.z = vh * u;
    F.w = 0;
    return F;
}

__host__ __device__ gridpoint G(gridpoint gp)
{
    float h = gp.x;
    float uh = gp.y;
    float vh = gp.z;
    
    float h4 = h*h*h*h;
    float v = sqrtf(2)*h*vh/(sqrtf(h4 + max(h4, EPSILON)));

    gridpoint G;
    G.x = v*h;
    G.y = uh * v;
    G.z = vh * v + GRAVITY * h * h;
    G.w = 0;
    return G;
}

__host__ __device__ gridpoint H(gridpoint c, gridpoint n, gridpoint e, gridpoint s, gridpoint w)
{
    float h = c.x;

    gridpoint H;
    H.x = 0;
    H.y = -GRAVITY * h * (e.w-w.w);
    H.z = -GRAVITY * h * (s.w-n.w);
    H.w = 0;
    return H;
}

__host__ __device__ gridpoint operator +(const gridpoint& x, const gridpoint& y)
{
    gridpoint z;
    z.x = x.x + y.x;
    z.y = x.y + y.y;
    z.z = x.z + y.z;
    z.w = x.w + y.w;
    return z;
}
__host__ __device__ gridpoint operator -(const gridpoint& x, const gridpoint& y)
{
    gridpoint z;
    z.x = x.x - y.x;
    z.y = x.y - y.y;
    z.z = x.z - y.z;
    z.w = x.w - y.w;
    return z;
}
__host__ __device__ gridpoint operator *(const gridpoint& x, const gridpoint& y)
{
    gridpoint z;
    z.x = y.x * x.x;
    z.y = y.y * x.y;
    z.z = y.z * x.z;
    z.w = y.w * x.w;
    return z;
}
__host__ __device__ gridpoint operator *(const gridpoint& x, const float& c)
{
    gridpoint z;
    z.x = c * x.x;
    z.y = c * x.y;
    z.z = c * x.z;
    z.w = c * x.w;
    return z;
}
__host__ __device__ gridpoint operator *(const float& c, const gridpoint& x)
{
    return x * c;
}

__host__ __device__ void fixShore(gridpoint& l, gridpoint& c, gridpoint& r)
{
    if(r.x < 0.0f || l.x < 0.0f || c.x < 0.0f)
    {
        float h = r.w - l.w - c.w;
        l.x = 0.0f;
        r.x = 0.0f;
        c.x = max(h, 0.0f);
    }
    float h = c.x;
    float h4 = h*h*h*h;
    float v = sqrtf(2)*h*c.y/(sqrtf(h4 + max(h4, EPSILON)));
    float u = sqrtf(2)*h*c.z/(sqrtf(h4 + max(h4, EPSILON)));
    
    c.y = u*h;
    c.z = v*h;
}

__global__ void simulateWaveStep(gridpoint* grid_next, int width, int height, float timestep, int pitch)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if(x < width && y < height)
    {
        int gridx = x + 1;
        int gridy = y + 1;
        
        gridpoint center = tex2D(texture_grid, gridx, gridy);
        
        gridpoint north = tex2D(texture_grid, gridx, gridy - 1);
        
        gridpoint west = tex2D(texture_grid, gridx - 1, gridy);
        
        gridpoint south = tex2D(texture_grid, gridx, gridy + 1);
        
        gridpoint east = tex2D(texture_grid, gridx + 1, gridy);
        
        fixShore(west, center, east);
        fixShore(north, center, south);
        
        gridpoint u_south = 0.5f * ( south + center ) - timestep * ( G(south) - G(center) );
        gridpoint u_north = 0.5f * ( north + center ) - timestep * ( G(center) - G(north) );
        gridpoint u_west = 0.5f * ( west + center ) - timestep * ( F(center) - F(west) );
        gridpoint u_east = 0.5f * ( east + center ) - timestep * ( F(east) - F(center) );
        
        
        gridpoint u_center = center + timestep * H(center, north, east, south, west) - timestep *( F(u_east) - F(u_west) ) - timestep * ( G(u_south) - G(u_north) );
        if(u_center.x < 0)
        {
            u_center.x = 0.0f;
            //u_center.y = 0.0f;
            //u_center.z = 0.0f;
        }
        //u_center.x = max(0.0f, u_center.x);
        grid2Dwrite(grid_next, gridx, gridy, pitch, u_center);
    }
}

__global__ void initGrid(gridpoint *grid, int gridwidth, int gridheight, int pitch)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if(x < gridwidth && y < gridheight)
    {
        float a = tex2D(texture_landscape, x-1, y-1).y;
        
        gridpoint gp;
        gp.x = max(NN-a, 0.0f);
        gp.y = 0.0f;
        gp.z = 0.0f;
        gp.w = a;
        grid2Dwrite(grid, x, y, pitch, gp);
    }
}

__host__ __device__ vertex gridpointToVertex(gridpoint gp, float x, float y)
{
    float h = gp.x;
    if(h < 0.1f)
        h = -0.001f;
    vertex v;
    v.x = x * 20.0f - 10.0f;
    v.z = y * 20.0f - 10.0f;
    v.y = h+gp.w;
    return v;
}

__host__ __device__ rgb gridpointToColor(gridpoint gp)
{
    rgb c;
    c.x = min(20 + (gp.x+gp.w - NN) / (NN / 10) * 150.0f, 255);
    c.y = min(40 + (gp.x+gp.w - NN) / (NN / 10) * 150.0f, 255);
    c.z = min(100 + (gp.x+gp.w - NN) / (NN / 10) * 150.0f, 255);
    c.w = 255-max(-50*gp.x+50, 0);
    return c;
}

__global__ void visualise(vertex* watersurfacevertices,
                          rgb* watersurfacecolors, int width, int height)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if(x < width && y < height)
    {
        int gridx = x + 1;
        int gridy = y + 1;
        
        gridpoint gp = tex2D(texture_grid, gridx, gridy);

        watersurfacevertices[y * width + x] = gridpointToVertex(gp, x / float(width - 1), y / float(height - 1));
        watersurfacecolors[y * width + x] = gridpointToColor(gp);
    }
}


__global__ void addWave(gridpoint* grid, float* wave, float norm, int width, int height, int pitch)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if(x < width && y < height)
    {
        int gridx = x + 1;
        int gridy = y + 1;

        float waveheight = grid2Dread(grid, gridx, gridy, pitch).x;

        waveheight += (grid2Dread(wave, x, y, width) - 5.0f) / norm;

        grid[ gridx + gridy * pitch ].x = waveheight;

    }
}

void addWave(float* wave, float norm, int width, int height, int pitch_elements)
{
    hipError_t error;
    size_t sizeInBytes = width * height * sizeof(float);

    error = hipMemcpy(device_waves, wave, sizeInBytes, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    int x = (width + BLOCKSIZE_X - 1) / BLOCKSIZE_X;
    int y = (height + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y;
    dim3 threadsPerBlock(BLOCKSIZE_X, BLOCKSIZE_Y);
    dim3 blocksPerGrid(x, y);

    addWave <<< blocksPerGrid, threadsPerBlock>>>(device_grid_next, device_waves, norm, width, height, pitch_elements);

    error = hipGetLastError();
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipDeviceSynchronize();
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    gridpoint *grid_helper = device_grid;
    device_grid = device_grid_next;
    device_grid_next = grid_helper;

    error = hipBindTexture2D(0, &texture_grid, device_grid, &grid_channeldesc, width + 2, height + 2, grid_pitch_elements * sizeof(gridpoint));
    CHECK_EQ(hipSuccess, error) << "Error at line " << __LINE__ << ": " << hipGetErrorString(error);
}

void initWaterSurface(int width, int height, vertex *heightmapvertices, float *wave)
{

    if(state != UNINTIALISED)
    {
        return;
    }
    stepsperframe = FLAGS_wspf;
    timestep = FLAGS_timestep;
    int gridwidth = width + 2;
    int gridheight = height + 2;

    size_t sizeInBytes;
    size_t grid_pitch;
    hipError_t error;

    grid_channeldesc = hipCreateChannelDesc<float4>();
    hipChannelFormatDesc treshholds_channeldesc = hipCreateChannelDesc<float>();
    hipChannelFormatDesc reflections_channeldesc = hipCreateChannelDesc<int>();

    //alloc pitched memory for device_grid
    error = hipMallocPitch(&device_grid, &grid_pitch, gridwidth * sizeof(gridpoint), gridheight);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    CHECK_NOTNULL(device_grid);

    size_t oldpitch = grid_pitch;

    //alloc pitched memoty for device_grid_next
    error = hipMallocPitch(&device_grid_next, &grid_pitch, gridwidth * sizeof(gridpoint), gridheight);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    CHECK_NOTNULL(device_grid_next);

    CHECK_EQ(oldpitch, grid_pitch);

    grid_pitch_elements = grid_pitch / sizeof(gridpoint);

    //alloc pitched memory for landscape data on device
    size_t heightmap_pitch;
    error = hipMallocPitch(&device_heightmap, &heightmap_pitch, width * sizeof(vertex), height);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    CHECK_NOTNULL(device_heightmap);

    // copy landscape data to device
    error = hipMemcpy2D(device_heightmap, heightmap_pitch, heightmapvertices, width * sizeof(vertex), width * sizeof(vertex), height, hipMemcpyHostToDevice);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    
    // bind heightmap to texture_landscape
    hipChannelFormatDesc heightmap_channeldesc = hipCreateChannelDesc<float4>();
    error = hipBindTexture2D(0, &texture_landscape, device_heightmap, &heightmap_channeldesc, width, height, heightmap_pitch);
    CHECK_EQ(hipSuccess, error) << "Error at line " << __LINE__ << ": " << hipGetErrorString(error);
    
    // malloc memory for watersurface vertices
    sizeInBytes = width*height*sizeof(vertex);
    error = hipMalloc(&device_watersurfacevertices, sizeInBytes);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    // malloc memory for watersurface colors
    sizeInBytes = height * width * sizeof(rgb);
    error = hipMalloc(&device_watersurfacecolors, sizeInBytes);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    // malloc memory for waves
    sizeInBytes = height * width * sizeof(float);
    error = hipMalloc(&device_waves, sizeInBytes);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    // make dimension
    int x = (gridwidth + BLOCKSIZE_X - 1) / BLOCKSIZE_X;
    int y = (gridheight + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y;
    dim3 threadsPerBlock(BLOCKSIZE_X, BLOCKSIZE_Y);
    dim3 blocksPerGrid(x, y);

    int x1 = (gridwidth + BLOCKSIZE_X - 1) / BLOCKSIZE_X;
    dim3 threadsPerBlock1(BLOCKSIZE_X, 1);
    dim3 blocksPerGrid1(x1, 1);

    int y1 = (gridheight + BLOCKSIZE_Y -  1) / BLOCKSIZE_Y;
    dim3 threadsPerBlock2(1, BLOCKSIZE_Y);
    dim3 blocksPerGrid2(1, y1);

    //init grid with initial values
    initGrid <<< blocksPerGrid, threadsPerBlock>>>(device_grid, gridwidth, gridheight, grid_pitch_elements);

    //init grid_next with initial values
    initGrid <<< blocksPerGrid, threadsPerBlock>>>(device_grid_next, gridwidth, gridheight, grid_pitch_elements);

    error = hipDeviceSynchronize();
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);


    //bind the grid to texture_grid
    error = hipBindTexture2D(0, &texture_grid, device_grid, &grid_channeldesc, gridwidth, gridheight, grid_pitch);
    CHECK_EQ(hipSuccess, error) << "Error at line " << __LINE__ << ": " << hipGetErrorString(error);

    //add the initial wave to the grid
    addWave(wave, 0.5f, width, height, grid_pitch_elements);

    state = INITIALISED;
}

void computeNext(int width, int height, vertex* watersurfacevertices, rgb* watersurfacecolors)
{
    if(state != INITIALISED)
    {
        return;
    }

    int gridwidth = width + 2;
    int gridheight = height + 2;

    hipError_t error;
    // make dimension
    int x = (gridwidth + BLOCKSIZE_X - 1) / BLOCKSIZE_X;
    int y = (gridheight + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y;
    dim3 threadsPerBlock(BLOCKSIZE_X, BLOCKSIZE_Y);
    dim3 blocksPerGrid(x, y);

    //gitter "stepsperframe" zeitschritt
    for(int x = 0; x < stepsperframe; x++)
    {
        simulateWaveStep <<< blocksPerGrid, threadsPerBlock>>>(device_grid_next, width, height, timestep, grid_pitch_elements);

        error = hipDeviceSynchronize();
        CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

        gridpoint *grid_helper = device_grid;
        device_grid = device_grid_next;
        device_grid_next = grid_helper;

        error = hipBindTexture2D(0, &texture_grid, device_grid, &grid_channeldesc, gridwidth, gridheight, grid_pitch_elements * sizeof(gridpoint));
        CHECK_EQ(hipSuccess, error) << "Error at line " << __LINE__ << ": " << hipGetErrorString(error);
    }
    visualise <<< blocksPerGrid, threadsPerBlock >>>(device_watersurfacevertices, device_watersurfacecolors, width, height);

    error = hipGetLastError();
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipDeviceSynchronize();
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);

    // copy back data
    error = hipMemcpy(watersurfacevertices, device_watersurfacevertices, width * height * sizeof(vertex), hipMemcpyDeviceToHost);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipMemcpy(watersurfacecolors, device_watersurfacecolors, width * height * sizeof(rgb), hipMemcpyDeviceToHost);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
}

void destroyWaterSurface()
{
    if(state != INITIALISED)
    {
        return;
    }

    state = UNINTIALISED;
}
