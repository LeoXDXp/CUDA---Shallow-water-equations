#include "hip/hip_runtime.h"
#include "wavesimulator.h"

#include <glog/logging.h>

#include "types.h"

const int UNINTIALISED = 0;
const int INITIALISED = 1;
const float GRAVITY = 9.8;

gridpoint* device_grid;
vertex* device_heightmap;
vertex* device_watersurfacevertices;
rgb* device_watersurfacecolors;

int f;


int state = UNINTIALISED;
/*
__host__ __device__ float3 U(float h, velocity v)
{
    float3 U;
    U.x = h;
    U.y = v.x * h;
    U.z = v.y * h;
    return U;
}

__host__ __device__ float3 F(float h, velocity v)
{
    float3 F;
    F.x = v.x * h;
    F.y = (v.x * v.x * h) + ((1 / 2) * GRAVITY * h * h);
    F.z = v.x * v.y * h;
    return F;
}

__host__ __device__ float3 G(float h, velocity v)
{
    float3 G;
    G.x = v.y * h;
    G.y = v.x * v.y * h;
    G.z = (v.y * v.y * h) + ((1 / 2) * GRAVITY * h * h);
    return G;
}
*/
#if __GPUVERSION__
__global__ void simulateWaveStep(int frame, gridpoint* device_grid, vertex* device_heightmap, 
                            vertex* device_watersurfacevertices, rgb* device_watersurfacecolors, 
                            int width, int height)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int gridx = x + 1;
    int gridy = y + 1;
    
    if(x < width && y < height)
    {
       vertex v;
       v.x = x/float(width)*16-8;
       v.z = y/float(height)*16-8;
       v.y = 0.05 * sin(frame / 10.0f + v.x * 20 - 5 * v.z) + 0.9;
       device_watersurfacevertices[y*width+x] = v;
       
       rgb c;
       c.x = 100 + 50 * (v.y - 0.9) * 20;
       c.y = 150 + 50 * (v.y - 0.9) * 20;
       c.z = 255;
       device_watersurfacecolors[y * width + x] = c;
	}
}
#endif
void initWaterSurface(int width, int height, vertex* heightmapvertices)
{

    if(state != UNINTIALISED)
    {
        return;
    }
#if __GPUVERSION__
    size_t sizeInBytes;
    hipError_t error;
          
    // malloc memory for device_grid
    sizeInBytes = (height+2) * (width+2) * sizeof(gridpoint);
    error = hipMalloc(&device_grid, sizeInBytes);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    
    // malloc memory for device_watersurfacevertices
    sizeInBytes = height * width * sizeof(vertex);
    error = hipMalloc(&device_watersurfacevertices, sizeInBytes);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    
    // malloc memory for watersurfacecolors
    sizeInBytes = height * width * sizeof(rgb);
    error = hipMalloc(&device_watersurfacecolors, sizeInBytes);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    
    // copy heightmapdata data to device
    sizeInBytes = height * width * sizeof(vertex);
    error = hipMalloc(&device_heightmap, sizeInBytes);
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
     
#endif
    
    state = INITIALISED;
}

void computeNext(float time, int width, int height, vertex* watersurfacevertices, rgb* watersurfacecolors)
{
    if(state != INITIALISED)
    {
        return;
    }
    #if __GPUVERSION__
    hipError_t error;
    
     // make dimension
    int x = (width + 16 - 1) / 16;
    int y = (height + 16 - 1) / 16;
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid(x, y);
    
    //gitter 1 zeitschritt
    simulateWaveStep<<<blocksPerGrid, threadsPerBlock>>>(f++ ,device_grid, device_heightmap, device_watersurfacevertices, 
                     device_watersurfacecolors, width, height);
    
    error = hipGetLastError();
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    error = hipDeviceSynchronize();
    CHECK_EQ(hipSuccess, error) << "Error: " << hipGetErrorString(error);
    

    
    // copy back data
    hipMemcpy(watersurfacevertices, device_watersurfacevertices, width*height*sizeof(vertex), hipMemcpyDeviceToHost);
    hipMemcpy(watersurfacecolors, device_watersurfacecolors, width*height*sizeof(rgb), hipMemcpyDeviceToHost);
    #endif
}

void destroyWaterSurface()
{
    if(state != INITIALISED)
    {
        return;
    }

    state = UNINTIALISED;
}

















